#include <pgc/System.h>

int main(int argc, char **argv)
{
   Pscf::Pspg::Continuous::System<1> system;

   // Process command line options
   system.setOptions(argc, argv);

   // Read parameters from default parameter file
   system.readParam();

   // Read command script to run system
   system.readCommands();
   // hipDeviceReset();
   return 0;
}
