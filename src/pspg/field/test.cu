#include "hip/hip_runtime.h"
#include "FCT.h"

#include <iomanip>
#include<time.h>

// 3-D
/*
int main()
{
    FCT<3> fct;
    int mesh[3];
    int Nx = 4,
        Ny = 4,
        Nz = 8;
    int size = Nx * Ny * Nz;
    mesh[0] = Nx;
    mesh[1] = Ny;
    mesh[2] = Nz;
    double *data_c, *data;
    data_c = new double[size];
    hipMalloc((void**)&data, size * sizeof(double));

    srand((unsigned int)time(NULL));
    for(int i = 0; i < size; ++i)
        // data_c[i] = 1.0;
        data_c[i] = rand()%10;
    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    std::cout << "----------------------------------------------------------------" << std::endl;

    hipMemcpy(data, data_c, size * sizeof(double), hipMemcpyHostToDevice);

    fct.setup(mesh);
    fct.forwardTransform(data);

    hipMemcpy(data_c, data, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    std::cout << "----------------------------------------------------------------" << std::endl;
    fct.inverseTransform(data);

    hipMemcpy(data_c, data, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }

    free(data_c);
    hipFree(data);

    return 0;
}
*/

// 2-D
/*int main()
{
    int mesh[3], perm[3];
    int Nz = 6,
        Ny = 4;
    int size = Nz * Ny;
    mesh[0] = 1;
    mesh[1] = Ny;
    mesh[2] = Nz;

    perm[0] = 1;
    perm[1] = 0;
    perm[2] = 2;

    hipfftDoubleReal *in_d, *out_d, *in_c, *out_c;
    in_c  = new hipfftDoubleReal[size];
    out_c = new hipfftDoubleReal[size];
    hipMalloc((void**)&in_d,  size * sizeof(hipfftDoubleReal));
    hipMalloc((void**)&out_d, size * sizeof(hipfftDoubleReal));

    srand((unsigned int)time(NULL));
    for(int i = 0; i < size; ++i)
        in_c[i] = rand()%10;
    
    for(int y = 0; y < Ny; ++y)
    {
        for(int z = 0; z < Nz; ++z)
            std::cout << in_c[z + Nz*y] << "   ";
        std::cout << "\n";
    }
    std::cout << "\n";

    hipMemcpy(in_d, in_c, size*sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);

    cut_transpose3d( out_d,
                     in_d,
                     mesh,
                     perm,
                     1);
    hipMemcpy(out_c, out_d, size*sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);    

    for(int y = 0; y < Nz; ++y)
    {
        for(int z = 0; z < Ny; ++z)
        {
            std::cout << out_c[y+Nz*z] << "   ";
            // std::cout << y+Ny*x+1 << "\n";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
    mesh[0] = 1;
    mesh[1] = Nz;
    mesh[2] = Ny;

    perm[0] = 2;
    perm[1] = 1;
    perm[2] = 0;

    cut_transpose3d( in_d,
                     out_d,
                     mesh,
                     perm,
                     1);
    hipMemcpy(out_c, in_d, size*sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);    

    for(int y = 0; y < Ny; ++y)
    {
        for(int z = 0; z < Nz; ++z)
        {
            std::cout << out_c[y+Ny*z] << "   ";
            // std::cout << y+Ny*x+1 << "\n";
        }
        std::cout << "\n";
    }

    delete [] in_c; 
    delete [] out_c;
    hipFree(in_d);
    hipFree(out_d);


   FCT<2> fct;
    int mesh[3];
    int Nx = 1,
        Ny = 4,
        Nz = 4;
    int size = Nx * Ny * Nz;
    mesh[0] = Nx;
    mesh[1] = Ny;
    mesh[2] = Nz;
    double *data_c, *data;
    data_c = new double[size];
    hipMalloc((void**)&data, size * sizeof(double));

    srand((unsigned int)time(NULL));
    for(int i = 0; i < size; ++i)
        // data_c[i] = 1.0;
        data_c[i] = rand()%10;
    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                          << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    std::cout << "----------------------------------------------------------------" << std::endl;

    hipMemcpy(data, data_c, size * sizeof(double), hipMemcpyHostToDevice);

    fct.setup(mesh);
    fct.forwardTransform(data);
    // exit(1);

    hipMemcpy(data_c, data, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    std::cout << "----------------------------------------------------------------" << std::endl;
    fct.inverseTransform(data);

    hipMemcpy(data_c, data, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }

    free(data_c);
    hipFree(data);

    return 0;

}*/

int main()
{
    FCT<1> fct;
    int mesh[1];
    int Nx = 1,
        Ny = 1,
        Nz = 16;
    int size = Nx * Ny * Nz;
    mesh[0] = Nx;
    mesh[1] = Ny;
    mesh[2] = Nz;
    double *data_c, *data;
    data_c = new double[size];
    hipMalloc((void**)&data, size * sizeof(double));

    srand((unsigned int)time(NULL));
    for(int i = 0; i < size; ++i)
        data_c[i] = std::cos(M_PI*(i+0.5)/size);
    
        // data_c[i] = rand()%10;
    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    std::cout << "----------------------------------------------------------------" << std::endl;

    hipMemcpy(data, data_c, size * sizeof(double), hipMemcpyHostToDevice);

    fct.setup(mesh);
    fct.forwardTransform(data);

    hipMemcpy(data_c, data, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    std::cout << "----------------------------------------------------------------" << std::endl;
    fct.inverseTransform(data);

    hipMemcpy(data_c, data, size * sizeof(double), hipMemcpyDeviceToHost);

    for (int z = 0; z < Nz; z++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int x = 0; x < Nx; x++)
            {
                std::cout << std::setw(8) << std::scientific
                << data_c[z + y*Nz + Nz*Ny*x] << "   ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }

    free(data_c);
    hipFree(data);

    return 0;
}
