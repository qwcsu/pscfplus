#include "hip/hip_runtime.h"
#ifndef PSPG_LINEAR_ALGEBRA_CU
#define PSPG_LINEAR_ALGEBRA_CU

#include "LinearAlgebra.h"

namespace Pscf
{
   namespace Pspg
   {

      __global__ void subtractUniform(cudaReal *result, cudaReal rhs, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] -= rhs;
         }
      }

      __global__ void addUniform(cudaReal *result, cudaReal rhs, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] += rhs;
         }
      }

      __global__ void pointWiseSubtract(cudaReal *result, const cudaReal *rhs, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] -= rhs[i];
         }
      }

      __global__ void pointWiseSubtractFloat(cudaReal *result, const float rhs, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] -= rhs;
         }
      }

      __global__ void pointWiseBinarySubtract(const cudaReal *a, const cudaReal *b, cudaReal *result, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] = a[i] - b[i];
         }
      }

      __global__ void pointWiseAdd(cudaReal *result, const cudaReal *rhs, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] += rhs[i];
         }
      }

      __global__ void pointWiseBinaryAdd(const cudaReal *a, const cudaReal *b, cudaReal *result, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] = a[i] + b[i];
         }
      }

      __global__ void pointWiseAddScale(cudaReal *result, const cudaReal *rhs, double scale, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] += scale * rhs[i];
         }
      }

      __global__ void pointWiseAddScale2(cudaReal *result, const cudaReal *rhs, const cudaReal *rhs2, double scale, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] += scale * rhs[i] * rhs2[i];
         }
      }

      __global__ void inPlacePointwiseMul(cudaReal *a, const cudaReal *b, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            a[i] *= b[i];
         }
      }

      __global__ void pointwiseMul(const cudaReal *a, const cudaReal *b, cudaReal *result, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] = a[i] * b[i];
         }
      }

      __global__ void assignUniformReal(cudaReal *result, cudaReal uniform, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] = uniform;
         }
      }

      __global__ void assignReal(cudaReal *result, const cudaReal *rhs, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] = rhs[i];
         }
      }

      __global__ void assignExp(cudaReal *out, const cudaReal *w, double constant, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            out[i] = exp(-w[i] * constant);
         }
      }

      __global__ void scaleReal(cudaReal *result, double scale, int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;

         for (int i = startID; i < size; i += nThreads)
         {
            result[i] *= scale;
         }
      }

      __global__ void cudaComplexMulAdd(cudaReal *result,
                                        const cudaComplex *c1,
                                        const cudaComplex *c2,
                                        const cudaReal scale,
                                        int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            result[i] += scale * hipCmul(c1[i], hipConj(c2[i])).x;
         }
      }

      __global__ void cudaComplexAdd(cudaComplex *result,
                                     const cudaComplex *c1,
                                     int size)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < size; i += nThreads)
         {
            if (i == 0)
            {
               result[i].x += (c1[i].x - 1.0);
               result[i].y += c1[i].y;
            }
            else
            {
               result[i].x += c1[i].x;
               result[i].y += c1[i].y;
            }
         }
      }

      __global__ void mStressHelperIncmp(cudaReal *result,
                                         const cudaReal *c,
                                         const cudaReal *dksq,
                                         const cudaReal *dbu0K,
                                         int paramN,
                                         int kSize,
                                         int rSize)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < kSize; i += nThreads)
         {

            result[i] = dbu0K[i] * dksq[paramN * rSize + i] * c[i];
         }
      }

      __global__ void mStressHelper(cudaReal *result,
                                    const cudaReal *c,
                                    const cudaComplex *k1,
                                    const cudaReal *dksq,
                                    const cudaReal *dbu0,
                                    cudaReal kappaN,
                                    int paramN,
                                    int kSize,
                                    int rSize)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < kSize; i += nThreads)
         {

            result[i] = dbu0[i] * dksq[paramN * rSize + i] * (kappaN * (k1[i].x * k1[i].x - k1[i].y * k1[i].y) + c[i]);
         }
      }

      __global__ void sVHelper(cudaReal *result,
                               const cudaReal *rhoJ,
                               int nx)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < nx; i += nThreads)
         {
            result[i] = rhoJ[i]*log(abs(rhoJ[i]));
         }
      }

      __global__ void sSHelper(cudaReal *result,
                               const cudaReal *rhoS,
                               const cudaReal *w,
                               const cudaReal Q,
                               int nx)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < nx; i += nThreads)
         {
            result[i] = rhoS[i]*log(Q)+rhoS[i]*w[i];
         }
      }

      __global__ void sBlockHelper(cudaReal *result,
                                   const cudaReal *rhoJ,
                                   const cudaReal *q,
                                   int nx)
      {
         int nThreads = blockDim.x * gridDim.x;
         int startID = blockIdx.x * blockDim.x + threadIdx.x;
         for (int i = startID; i < nx; i += nThreads)
         {
            result[i] = rhoJ[i]*log(abs(q[i]));
         }
      }

   }
}
#endif
